#include "hip/hip_runtime.h"
#include <gmp.h>
#include <hip/hip_runtime.h>
#include "mp/mp.h"
#include "mp/gmp_conversion.h"
#include "test/testutil.h"


__global__
void cuda_mp_gt(mp_t r, const mp_t a, const mp_t b) {
	r[0] = mp_gt(a, b);
	return;
}

__global__
void cuda_mp_print(mp_p dev_a) {
	mp_print(dev_a);
}

int test() {
	gmp_randstate_t rand;
	gmp_randinit_default(rand);

	mpz_t gmp_a, gmp_b, gmp_r, gmp_r2;
	mpz_init(gmp_a);
	mpz_init(gmp_b);
	mpz_init(gmp_r);
	mpz_init(gmp_r2);

	mp_t a, b, r;

	mp_p dev_a;
	mp_dev_init(&dev_a);
	mp_p dev_b;
	mp_dev_init(&dev_b);
	mp_p dev_r;
	mp_dev_init(&dev_r);


	for (int i = 0; i < TEST_RUNS; i++) {
		mpz_urandomb(gmp_a, rand, BITWIDTH - 1);
		mpz_urandomb(gmp_b, rand, BITWIDTH - 1);

		mpz_to_mp(a, gmp_a);
		mpz_to_mp(b, gmp_b);

		mp_copy_to_dev(dev_a, a);
		mp_copy_to_dev(dev_b, b);

#ifdef LOG_LEVEL_VERBOSE_ENABLED
		printf("host:\n");
		mp_print(a);
		mp_print(b);

		printf("device:\n");
		cuda_mp_print<<<1,1>>>(dev_a);
		cuda_mp_print<<<1,1>>>(dev_b);
#endif

		cuda_mp_gt << < 1, 1 >> > (dev_r, dev_a, dev_b);

		mp_copy_from_dev(r, dev_r);

		if ((mpz_cmp(gmp_a, gmp_b) > 0) != r[0]) {
			printf("Comparison (gt) Test failed: %d  %d\n", (mpz_cmp(gmp_a, gmp_b)), r[0]);
			TEST_FAILURE;
		}
	}
	printf("Comparison (gt) Test Passed.\n");
	TEST_SUCCESS;
}

TEST_MAIN;
